#include "hip/hip_runtime.h"
#include "Data_kernel.cuh"

#define BLOCK_SIZE 32


__device__
bool Data_Push(
    int *weightFrom,
    int *weightTo,
    int *capacityFrom,
    int *capacityTo,
    int *heightFrom,
    int *heightTo
)
{
    if (*heightFrom != *heightTo + 1) {
        return false;
    }

    int value = min(*weightFrom, *capacityFrom);

    *weightFrom -= value;
    *capacityFrom -= value;
    *weightTo += value;
    *capacityTo += value;

    *heightTo = (*capacityTo) > 0 ? 1 : 0;
    return value > 0; // active
}


__device__
bool Data_PushLeft(
    int *device_weightLeft,
    int *device_weightRight,
    int *device_height,
    int *device_capacity,
    int columnSize,
    // parameters
    int x,
    int y
)
{
    if (x == 0) {
        return false;
    }

    const size_t indexFrom = x * columnSize + y;
    const size_t indexTo = (x - 1) * columnSize + y;

    return Data_Push(
        &device_weightLeft[indexFrom],
        &device_weightRight[indexTo],
        &device_capacity[indexFrom],
        &device_capacity[indexTo],
        &device_height[indexFrom],
        &device_height[indexTo]
    );
}


__device__
bool Data_PushRight(
    int *device_weightLeft,
    int *device_weightRight,
    int *device_height,
    int *device_capacity,
    int rowSize,
    int columnSize,
    // parameters
    int x,
    int y
)
{
    if (x == rowSize - 1) {
        return false;
    }

	const size_t indexFrom = x * columnSize + y;
	const size_t indexTo = (x + 1) * columnSize + y;

    return Data_Push(
        &device_weightRight[indexFrom],
        &device_weightLeft[indexTo],
        &device_capacity[indexFrom],
        &device_capacity[indexTo],
        &device_height[indexFrom],
        &device_height[indexTo]
    );
}


__device__
bool Data_PushUp(
    int *device_weightUp,
    int *device_weightDown,
    int *device_height,
    int *device_capacity,
    int columnSize,
    // parameters
    int x,
    int y
)
{
	if (y == columnSize - 1) {
		return false;
	}

	const size_t indexFrom = x * columnSize + y;
	const size_t indexTo = x * columnSize + (y + 1);

    return Data_Push(
        &device_weightUp[indexFrom],
        &device_weightDown[indexTo],
        &device_capacity[indexFrom],
        &device_capacity[indexTo],
        &device_height[indexFrom],
        &device_height[indexTo]
    );
}


__device__
bool Data_PushDown(
    int *device_weightUp,
    int *device_weightDown,
    int *device_height,
    int *device_capacity,
    int columnSize,
    // parameters
    int x,
    int y
)
{
	if (y == 0) {
		return false;
	}

	const size_t indexFrom = x * columnSize + y;
	const size_t indexTo = x * columnSize + (y - 1);

    return Data_Push(
        &device_weightDown[indexFrom],
        &device_weightUp[indexTo],
        &device_capacity[indexFrom],
        &device_capacity[indexTo],
        &device_height[indexFrom],
        &device_height[indexTo]
    );
}


__device__
void Data_PushFromS(
    int *device_weightS,
    int *device_height,
    int *device_capacity,
    int columnSize,
    int x,
    int y
)
{
	const size_t index = x * columnSize + y;
	if (device_weightS[index] > 0) {
		device_height[index] = 1;
	}
	device_capacity[index] += device_weightS[index];
	//this->weightS[index] = 0;
}


__device__
void Data_PushToT(
    int *device_weightT,
    int *device_height,
    int *device_capacity,
    int columnSize,
    int x,
    int y
)
{
	const size_t index = x * columnSize + y;
	int value = min(device_capacity[index], device_weightT[index]);
	device_capacity[index] -= value;
	//this->flow += value;
}


__global__
void Data_PushLeftForLine(
    bool *device_active,
    int *device_weightLeft,
    int *device_weightRight,
    int *device_weightS,
    int *device_weightT,
    int *device_height,
    int *device_capacity,
    int rowSize,
    int columnSize
)
{
    bool local_active = false;

    for (int i = BLOCK_SIZE - 1; i >= 1; i --) {
        int x = blockIdx.x * BLOCK_SIZE + i;
        int y = blockIdx.y * BLOCK_SIZE + threadIdx.x;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
            bool active = Data_PushLeft(
                device_weightLeft,
                device_weightRight,
                device_height,
                device_capacity,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;
            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }
    __syncthreads();
    { // i = 0
        int x = blockIdx.x * BLOCK_SIZE;
        int y = blockIdx.y * BLOCK_SIZE + threadIdx.x;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
            bool active = Data_PushLeft(
                device_weightLeft,
                device_weightRight,
                device_height,
                device_capacity,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;
            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }

    if (local_active) {
        *device_active = true;
    }
}


__global__
void Data_PushRightForLine(
    bool *device_active,
    int *device_weightLeft,
    int *device_weightRight,
    int *device_weightS,
    int *device_weightT,
    int *device_height,
    int *device_capacity,
    int rowSize,
    int columnSize
)
{
    bool local_active = false;

    for (int i = 0; i < BLOCK_SIZE - 1; i ++) {
        int x = blockIdx.x * BLOCK_SIZE + i;
        int y = blockIdx.y * BLOCK_SIZE + threadIdx.x;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
            bool active = Data_PushRight(
                device_weightLeft,
                device_weightRight,
                device_height,
                device_capacity,
                rowSize,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;
            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }
    __syncthreads();
    { // i = BLOCK_SIZE - 1
        int x = blockIdx.x * BLOCK_SIZE + (BLOCK_SIZE - 1);
        int y = blockIdx.y * BLOCK_SIZE + threadIdx.x;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
            bool active = Data_PushRight(
                device_weightLeft,
                device_weightRight,
                device_height,
                device_capacity,
                rowSize,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;
            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }

    if (local_active) {
        *device_active = true;
    }
}


__global__
void Data_PushUpForLine(
    bool *device_active,
    int *device_weightUp,
    int *device_weightDown,
    int *device_weightS,
    int *device_weightT,
    int *device_height,
    int *device_capacity,
    int rowSize,
    int columnSize
)
{
    bool local_active = false;

    for (int i = 0; i < BLOCK_SIZE - 1; i ++) {
        int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        int y = blockIdx.y * BLOCK_SIZE + i;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );

            bool active = Data_PushUp(
                device_weightUp,
                device_weightDown,
                device_height,
                device_capacity,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;

            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }
    __syncthreads();
    { // i = BLOCK_SIZE - 1
        int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        int y = blockIdx.y * BLOCK_SIZE + (BLOCK_SIZE - 1);
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );

            bool active = Data_PushUp(
                device_weightUp,
                device_weightDown,
                device_height,
                device_capacity,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;

            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }

    if (local_active) {
        *device_active = true;
    }
}


__global__
void Data_PushDownForLine(
    bool *device_active,
    int *device_weightUp,
    int *device_weightDown,
    int *device_weightS,
    int *device_weightT,
    int *device_height,
    int *device_capacity,
    int rowSize,
    int columnSize
)
{
    bool local_active = false;

    for (int i = BLOCK_SIZE - 1; i >= 1; i --) {
        int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        int y = blockIdx.y * BLOCK_SIZE + i;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );

            bool active = Data_PushDown(
                device_weightUp,
                device_weightDown,
                device_height,
                device_capacity,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;

            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }
    __syncthreads();
    { // i = 0
        int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        int y = blockIdx.y * BLOCK_SIZE;
        if (x < rowSize && y < columnSize) {
            Data_PushFromS(
                device_weightS,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );

            bool active = Data_PushDown(
                device_weightUp,
                device_weightDown,
                device_height,
                device_capacity,
                columnSize,
                // parameters
                x,
                y
            );
            local_active = local_active || active;

            Data_PushToT(
                device_weightT,
                device_height,
                device_capacity,
                columnSize,
                x,
                y
            );
        }
    }

    if (local_active) {
        *device_active = true;
    }
}


__global__
void Data_BfsFromT(
    int *device_weightT,
    int *device_height,
    int rowSize,
    int columnSize
)
{
    int tid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.y + threadIdx.x;
    //int tid = threadIdx.y * blockDim.y + threadIdx.x;

    if (tid < rowSize * columnSize) {
        int x = tid / columnSize, y = tid % columnSize;
        if (device_weightT[x * columnSize + y] > 0) {
            device_height[x * columnSize + y] = 1;
        } else {
            device_height[x * columnSize + y] = -1;
        }
    }
}


__global__
void Data_BfsLevelK(
    bool *device_active,
    int *device_weightUp,
    int *device_weightDown,
    int *device_weightLeft,
    int *device_weightRight,
    int *device_height,
    int rowSize,
    int columnSize,
    // parameter
    int k
)
{
    int tid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.y + threadIdx.x;

    if (tid < rowSize * columnSize) {
        int x = tid / columnSize, y = tid % columnSize;

        int centerIndex = x * columnSize + y;
        int leftIndex = (x - 1) * columnSize + y;
        int rightIndex = (x + 1) * columnSize + y;
        int upIndex = x * columnSize + (y + 1);
        int downIndex = x * columnSize + (y - 1);

        if ( device_height[centerIndex] == -1 && (
            ( x != 0 && device_height[leftIndex] == k && device_weightLeft[centerIndex] > 0 ) || // left
            ( x != rowSize - 1 && device_height[rightIndex] == k && device_weightRight[centerIndex] > 0 ) || // right
            ( y != columnSize - 1 && device_height[upIndex] == k && device_weightUp[centerIndex] > 0 ) || // up
            ( y != 0 && device_height[downIndex] == k && device_weightDown[centerIndex] > 0 ) // down
        ) ) {
            device_height[centerIndex] = k + 1;
            *device_active = true;
        }
    }
}

